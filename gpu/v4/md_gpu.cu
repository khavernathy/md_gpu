#include "hip/hip_runtime.h"
/* ==================================================================
	Programmers: Alfredo Peguero Tejada & Douglas Franz
	A molecular dynamics NVE code for GPU.
	To compile: nvcc my_file.cu -o my_exe in the rc machines
	run with, e.g. ./my_exe 
   ==================================================================
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <sstream>
#include <algorithm>
#include <iterator>
#include <fstream>
#include <map>
#include "constants.cpp"
#include "system.cpp"
#include <vector>


/* descriptors for single atom in the tree */
typedef struct atomdesc {
	double px, py, pz, vx, vy, vz, ax, ay, az, fx, fy, fz, charge, mass, LJsig, LJeps;
	char name[2];
} atom;

atom * atom_list;		/* list of all data points  for GPU             */

// These are for an old way of tracking time 
struct timezone Idunno;	
struct timeval startTime, endTime;

//	set a checkpoint and show the (natural) running time in seconds 
double report_running_time() {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("Running time: %ld.%06ld s\n", sec_diff, usec_diff);
	printf("----------------------------------------------\n");
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}
/*
__global__
void newToOld(atom * new_list, atom * old_list, int n)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < n)
	{
		old_list[i] = new_list[i];
	}
}
*/

__global__
void calculateForce(atom * atom_list, double ts, int n)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < n) 
	{
		//printf("%le\n",atom_list[i].pz); //works

				// calculate forces
		// initialize to zero.
		atom_list[i].fx = 0.0;
		atom_list[i].fy = 0.0;
		atom_list[i].fz = 0.0;

		// loop through pairs
		for (int j=i+1; j<n; j++) 
		{
				//initialize atom j force = 0
				//atom_list[j].fx = 0.0;	
				//atom_list[j].fy = 0.0;
				//atom_list[j].fz = 0.0;
				
				// check mixing rules
				double eps = sqrt(atom_list[i].LJeps * atom_list[j].LJeps);
				double sig = 0.5*(atom_list[i].LJsig + atom_list[j].LJsig);

				// distances etc.
				double dx,dy,dz,rsq,r,ux,uy,uz,fx,fy,fz,ke;
				dx = atom_list[i].px - atom_list[j].px;
				dy = atom_list[i].py - atom_list[j].py;
				dz = atom_list[i].pz - atom_list[j].pz;

				rsq = dx*dx + dy*dy + dz*dz;
				r = sqrt(rsq);
				ux = dx/r;
				uy = dy/r;
				uz = dz/r;

				// LJ force
				fx = 24*dx*eps*(2*pow(sig,12)*pow(r,-14) - pow(sig,6)*pow(r,-8));
				fy = 24*dy*eps*(2*pow(sig,12)*pow(r,-14) - pow(sig,6)*pow(r,-8));
				fz = 24*dz*eps*(2*pow(sig,12)*pow(r,-14) - pow(sig,6)*pow(r,-8));
				
				atom_list[i].fx += fx;
				atom_list[i].fy += fy;// += fy;
				atom_list[i].fz +=fz; // += fz;
			
				atom_list[j].fx += fx; //atom_list[j].fx -= fx;
				atom_list[j].fy += fy; // -= fy;
				atom_list[j].fz += fz; // -= fz;	
				
				// electrostatic force
				ke = 8.987551787e9;
				fx = (ke * (atom_list[i].charge * atom_list[j].charge)/rsq) * ux;
                                fy = (ke * (atom_list[i].charge * atom_list[j].charge)/rsq) * uy;
                                fz = (ke * (atom_list[i].charge * atom_list[j].charge)/rsq) * uz;

                                atom_list[i].fx +=fx; // += fx;
                                atom_list[i].fy+= fy; // += fy;
                                atom_list[i].fz += fz; // += fz;

                                atom_list[j].fx+= fx; // -= fx;
                                atom_list[j].fy+= fy; // -= fy;
                                atom_list[j].fz+= fz; // -= fz;

		}


	}	
}

__global__
void useTheForce(atom * atom_list, double ts, int n) 
{
	// do velocity verlet on forces to get accel. / vel.
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < n)
	{
		double prev_ax,prev_ay,prev_az;
		prev_ax = atom_list[i].ax;
		prev_ay = atom_list[i].ay;
		prev_az = atom_list[i].az;

		// a = F/m
		atom_list[i].ax = atom_list[i].fx / atom_list[i].mass;
		atom_list[i].ay = atom_list[i].fy / atom_list[i].mass;
		atom_list[i].az = atom_list[i].fz / atom_list[i].mass;

		// new velocity
		atom_list[i].vx = atom_list[i].vx + 0.5*(atom_list[i].ax + prev_ax)*ts;
		atom_list[i].vy = atom_list[i].vy + 0.5*(atom_list[i].ay + prev_ay)*ts;
		atom_list[i].vz = atom_list[i].vz + 0.5*(atom_list[i].az + prev_az)*ts;
			
		// integrate
		//integrate(atom_list[i], atom_list[i], ts);			
		atom_list[i].px = atom_list[i].px + atom_list[i].vx * ts + 0.5 * atom_list[i].ax * ts *ts;
		atom_list[i].py = atom_list[i].py + atom_list[i].vy * ts + 0.5 * atom_list[i].ay * ts *ts;
		atom_list[i].pz = atom_list[i].pz + atom_list[i].vz * ts + 0.5 * atom_list[i].az * ts *ts;
		

	}
}

void write(atom * atom_list, double time, int c, int n) 
{
	ofstream myfile;
	myfile.open ("outfile.xyz", ios_base::app);
	time = time * 1.0e15; // from fs to s
	myfile << n;
	myfile << "\n Time: ";
	myfile << time;
	myfile << " fs -- step count: ";
	myfile << c;
	myfile << "\n";

	for (int i =0; i < n; i++) {
		myfile << atom_list[i].name;  //"H ";
		//myfile << atom_list[i].name;//[0];
		//myfile << atom_list[i].name[1]; 
		//printf("%s",atom_list[i].name);
		myfile << "  ";
		myfile << atom_list[i].px*1e10;
		myfile << "  ";
		myfile << atom_list[i].py*1e10;
		myfile << "  ";
		myfile << atom_list[i].pz*1e10;
		myfile << "\n";
	} 
	myfile.close();
}

void runMD(atom * atom_list, int n, float ts, float tf) { 

	//printf("%s",atom_list[72].name);
	//printf("%le",atom_list[72].pz);

	int block_size = 32;

	// define memory requirements for atoms/histogram datasets.
	int atoms_size = n * sizeof(atom); 

        // write new device variable pointers
        atom *d_atom_list; // = atom_list;
	//atom *d_atom_list; 

	// allocate gpu memory and send data to gpu to old
        hipMalloc((void**) &d_atom_list, atoms_size);
        hipMemcpy(d_atom_list, atom_list, atoms_size, hipMemcpyHostToDevice);

	// and the new (duplicate)
	//hipMalloc((void**) &d_atom_list, atoms_size);
	//hipMemcpy(d_atom_list, atom_list, atoms_size, hipMemcpyHostToDevice);
	
		dim3 dimGrid(ceil(n/block_size),1,1);
		dim3 dimBlock(block_size,1,1);
	
		// time it
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord( start, 0 );
                	// go diego go
			int c = 0;
			for (float t=0.0; t <= tf; t+=ts) {
                		useTheForce<<< dimGrid, dimBlock >>>( d_atom_list, ts, n );
				calculateForce<<< dimGrid, dimBlock >>>( d_atom_list, ts, n );
				//newToOld<<< dimGrid, dimBlock >>>( d_atom_list, d_atom_list, n );
					
				// write to output file as needed.
				if (c%1 == 0)
				{
					// old and new are same at this point, so whatevs
					hipMemcpy(atom_list, d_atom_list, atoms_size, hipMemcpyDeviceToHost);
					//printf("%s",atom_list[0].px);
					write(atom_list, t, c, n);
				}
 
				c++; 
                	}
		// fetch kernel runtime
		hipEventRecord ( stop, 0 );
		hipEventSynchronize( stop );
		float elapsedTime;
		hipEventElapsedTime( &elapsedTime, start, stop );
		printf( "******** Total Running Time of doIt Kernel: %0.5f s ********\n", elapsedTime/1000.0 );
		hipEventDestroy( start );
		hipEventDestroy( stop );

	
	// all done. Free up device memory.
        hipFree(d_atom_list); //hipFree(d_atom_list);
}

void readFile(System &system, atom * atom_list) {
        //printf("%le",system.constants.kb);
	string line;
        ifstream myfile ("test2.dat"); // test2.dat
        if (myfile.is_open())
        {
                //std::string::size_type sz;     // alias of size_t
                // loop through each line
                int id = 0;
                while ( getline (myfile,line) )
                {
			vector<string> lc;
                        istringstream iss(line);
                        //ostream_iterator<string> out_it (cout,",");
                        copy(
                                istream_iterator<string>(iss),
                                istream_iterator<string>(),
                                back_inserter(lc) // "normally" out_it goes here.
                        );

			// make the atom from the current line.
			atom ca;		

			ca.name[0] = lc[0].c_str()[0];
			ca.name[1] = lc[0].c_str()[1];
			ca.px = atof(lc[1].c_str())* system.constants.cA;
			ca.py = atof(lc[2].c_str())* system.constants.cA;
			ca.pz = atof(lc[3].c_str())* system.constants.cA;
			ca.charge = atof(lc[4].c_str());
			ca.vx = 0.0;
			ca.vy = 0.0;
			ca.vz = 0.0;
			ca.ax = 0.0;
			ca.ay = 0.0;
			ca.az = 0.0;
			ca.fx = 0.0;
			ca.fy = 0.0;
			ca.fz = 0.0;
			ca.LJsig = system.constants.sigs[lc[0]];
			ca.LJeps = system.constants.eps[lc[0]];	
			ca.mass = system.constants.masses[lc[0]];

			//printf("%c%c %le %le %le %f %f %f %f %f %f %f %f %f %f %le %le %le\n",ca.name[0],ca.name[1], ca.px, ca.py, ca.pz, ca.charge, ca.vx, ca.vy, ca.vz, ca.ax, ca.ay, ca.az, ca.fx, ca.fy, ca.fz, ca.LJsig, ca.LJeps, ca.mass);		
			atom_list[id] = ca;
			id++;
		}
	}
}


//// MAIN =============================================================
int main(int argc, char **argv)
{

	// first delete outfile.xyz as needed.
	if ( remove( "outfile.xyz" ) != 0)
                perror( "Error deleting outfile.xyz" );
        else {
                cout << "outfile.xyz successfully deleted.";
                printf("\n");
        }

	int n = 75;
	
	System system; 
	float ts = 1.0e-15;
	float tf = 100e-15;
	
	// variable and memory assignments
	atom_list = (atom *)malloc(sizeof(atom)*n);
	//atom_list[200];

	// read da file which assigns atoms to atom_list
	readFile(system, atom_list);	

	//write(atom_list, 0, 0, n);
	
	//printf("%s",atom_list[0].name);
	
	// time the entire GPU process. 
	gettimeofday(&startTime, &Idunno);

	// run the function which calls the kernel, times the kernel, etc.
        runMD(atom_list, n, ts, tf); // uses same atom list as cpu code

	// spit back runtime.
        report_running_time();
	
	return 0;
}
